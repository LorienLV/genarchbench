#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "plink2_matrix_cuda.h"
#include <stdio.h>

int CudaGetDeviceCount() {
  int device_count;
  if (hipGetDeviceCount(&device_count) != hipSuccess) {
    return 0;
  }
  return device_count;
}

int CudaSetDevice(int device_idx) {
  if (hipSetDevice(device_idx) != hipSuccess) {
    return 1;
  }
  return 0;
}

void CublasFmultiplierPreinit(CublasFmultiplier* cfmp) {
  cfmp->handle = NULL;
  cfmp->dev_inmatrix1 = NULL;
  cfmp->dev_inmatrix2 = NULL;
  cfmp->dev_outmatrix = NULL;
  cfmp->in_borrowed = 0;
  cfmp->device_idx = -1;
}

void CublasFmultiplierBorrowColMajor1(CublasFmultiplier* srcp, CublasFmultiplier* dstp) {
  dstp->dev_inmatrix1 = srcp->dev_inmatrix1;
  dstp->in_borrowed |= 1;
}

void CublasFmultiplierBorrowColMajor2(CublasFmultiplier* srcp, CublasFmultiplier* dstp) {
  dstp->dev_inmatrix2 = srcp->dev_inmatrix2;
  dstp->in_borrowed |= 2;
}

// Initializes device handle and allocates device memory.
// Returns 0 on success, 1 on failure.
int CublasFmultiplierColMajorInit(int row1_ct, int col2_ct, int common_ct, CublasFmultiplier* cfmp) {
  if ((cfmp->device_idx != -1) || (cfmp->handle != NULL)) {
    // sanity check: improper function call
    return 1;
  }
  if (hipGetDevice(&cfmp->device_idx) != hipSuccess) {
    cfmp->device_idx = -1;
    return 1;
  }
  hipblasHandle_t* handlep = (hipblasHandle_t*)(&cfmp->handle);
  hipblasStatus_t stat = hipblasCreate(handlep);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    return 1;
  }
  if (!(cfmp->in_borrowed & 1)) {
    hipError_t cudaStat = hipMalloc((void**)&cfmp->dev_inmatrix1, row1_ct * sizeof(float) * common_ct);
    if (cudaStat != hipSuccess) {
      return 1;
    }
  }
  if (!(cfmp->in_borrowed & 2)) {
    hipError_t cudaStat = hipMalloc((void**)&cfmp->dev_inmatrix2, common_ct * sizeof(float) * col2_ct);
    if (cudaStat != hipSuccess) {
      return 1;
    }
  }
  hipError_t cudaStat = hipMalloc((void**)&cfmp->dev_outmatrix, row1_ct * sizeof(float) * col2_ct);
  if (cudaStat != hipSuccess) {
    return 1;
  }
  cfmp->row1_ct = row1_ct;
  cfmp->col2_ct = col2_ct;
  cfmp->common_ct = common_ct;
  return 0;
}

int CublasFmultiplierPreloadColMajor1(const float* inmatrix1, CublasFmultiplier* cfmp) {
  const int row1_ct = cfmp->row1_ct;
  hipblasStatus_t stat = hipblasSetMatrix(row1_ct, cfmp->common_ct, sizeof(float), inmatrix1, row1_ct, cfmp->dev_inmatrix1, row1_ct);
  return (stat != HIPBLAS_STATUS_SUCCESS);
}

int CublasFmultiplierPreloadColMajor2(const float* inmatrix2, CublasFmultiplier* cfmp) {
  const int common_ct = cfmp->common_ct;
  hipblasStatus_t stat = hipblasSetMatrix(common_ct, cfmp->col2_ct, sizeof(float), inmatrix2, common_ct, cfmp->dev_inmatrix2, common_ct);
  return (stat != HIPBLAS_STATUS_SUCCESS);
}

int CublasFmultiplyColMajor1(const float* inmatrix1, CublasFmultiplier* cfmp, float* outmatrix) {
  const int row1_ct = cfmp->row1_ct;
  const int col2_ct = cfmp->col2_ct;
  const int common_ct = cfmp->common_ct;
  hipblasStatus_t stat = hipblasSetMatrix(row1_ct, common_ct, sizeof(float), inmatrix1, row1_ct, cfmp->dev_inmatrix1, row1_ct);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    return 1;
  }
  float alpha = 1.0;
  float beta = 0.0;
  stat = hipblasSgemm((hipblasHandle_t)cfmp->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1_ct, col2_ct, common_ct, &alpha, cfmp->dev_inmatrix1, row1_ct, cfmp->dev_inmatrix2, common_ct, &beta, cfmp->dev_outmatrix, row1_ct);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    return 1;
  }
  stat = hipblasGetMatrix(row1_ct, col2_ct, sizeof(float), cfmp->dev_outmatrix, row1_ct, outmatrix, row1_ct);
  return (stat != HIPBLAS_STATUS_SUCCESS);
}

int CublasFmultiplyColMajor2(const float* inmatrix2, CublasFmultiplier* cfmp, float* outmatrix) {
  const int row1_ct = cfmp->row1_ct;
  const int col2_ct = cfmp->col2_ct;
  const int common_ct = cfmp->common_ct;
  hipblasStatus_t stat = hipblasSetMatrix(common_ct, col2_ct, sizeof(float), inmatrix2, common_ct, cfmp->dev_inmatrix2, common_ct);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    return 1;
  }
  float alpha = 1.0;
  float beta = 0.0;
  stat = hipblasSgemm((hipblasHandle_t)cfmp->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1_ct, col2_ct, common_ct, &alpha, cfmp->dev_inmatrix1, row1_ct, cfmp->dev_inmatrix2, common_ct, &beta, cfmp->dev_outmatrix, row1_ct);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    return 1;
  }
  stat = hipblasGetMatrix(row1_ct, col2_ct, sizeof(float), cfmp->dev_outmatrix, row1_ct, outmatrix, row1_ct);
  return (stat != HIPBLAS_STATUS_SUCCESS);
}

int CublasFmultiplierCleanup(CublasFmultiplier* cfmp) {
  if (cfmp->device_idx == -1) {
    return 0;
  }
  int retval = 0;
  if (hipSetDevice(cfmp->device_idx) != hipSuccess) {
    retval = 1;
  }
  cfmp->device_idx = -1;
  if (hipFree(cfmp->dev_outmatrix) != hipSuccess) {
    retval = 1;
  }
  cfmp->dev_outmatrix = NULL;
  if (!(cfmp->in_borrowed & 2)) {
    if (hipFree(cfmp->dev_inmatrix2) != hipSuccess) {
      retval = 1;
    }
  }
  cfmp->dev_inmatrix2 = NULL;
  if (!(cfmp->in_borrowed & 1)) {
    if (hipFree(cfmp->dev_inmatrix1) != hipSuccess) {
      retval = 1;
    }
  }
  cfmp->dev_inmatrix1 = NULL;
  cfmp->in_borrowed = 0;
  hipblasDestroy((hipblasHandle_t)cfmp->handle); // don't care about return value
  cfmp->handle = NULL;
  return retval;
}

// Only for testing purposes.
int CublasColMajorFmatrixMultiply(const float* inmatrix1, const float* inmatrix2, int row1_ct, int col2_ct, int common_ct, float* outmatrix) {
  CublasFmultiplier cfm;
  CublasFmultiplierPreinit(&cfm);
  int retval = 0;
  {
    if (CublasFmultiplierColMajorInit(row1_ct, col2_ct, common_ct, &cfm)) {
      goto CublasColMajorFmatrixMultiply_fail;
    }
    hipblasStatus_t stat = hipblasSetMatrix(row1_ct, common_ct, sizeof(float), inmatrix1, row1_ct, cfm.dev_inmatrix1, row1_ct);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      goto CublasColMajorFmatrixMultiply_fail;
    }
    stat = hipblasSetMatrix(common_ct, col2_ct, sizeof(float), inmatrix2, common_ct, cfm.dev_inmatrix2, common_ct);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      goto CublasColMajorFmatrixMultiply_fail;
    }
    float alpha = 1.0;
    float beta = 0.0;
    stat = hipblasSgemm((hipblasHandle_t)cfm.handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1_ct, col2_ct, common_ct, &alpha, cfm.dev_inmatrix1, row1_ct, cfm.dev_inmatrix2, common_ct, &beta, cfm.dev_outmatrix, row1_ct);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      goto CublasColMajorFmatrixMultiply_fail;
    }
    stat = hipblasGetMatrix(row1_ct, col2_ct, sizeof(float), cfm.dev_outmatrix, row1_ct, outmatrix, row1_ct);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      goto CublasColMajorFmatrixMultiply_fail;
    }
  }
  while (0) {
  CublasColMajorFmatrixMultiply_fail:
    retval = 1;
  }
  if (CublasFmultiplierCleanup(&cfm)) {
    retval = 1;
  }
  return retval;
}
